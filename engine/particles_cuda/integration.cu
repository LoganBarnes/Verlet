#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <stdio.h>

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>

#include "hip/hip_runtime_api.h"
#include "integration_kernel.cuh"
#include "util.cuh"
#include "shared_variables.cuh"

//#define PRINT

hiprandGenerator_t gen(0);

thrust::device_vector<float> V; // particle velocities
thrust::device_vector<float> lambda;
thrust::device_vector<float> denom;

thrust::device_vector<float> ros;

thrust::device_vector<uint> neighbors;
thrust::device_vector<uint> numNeighbors;

float *rands;

extern "C"
{
    /*****************************************************************************
     *                              INITIALIZATION
     *****************************************************************************/


    void initIntegration()
    {
        allocateArray((void **)&rands, 6 * sizeof(float));
        checkCudaErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
        checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
    }

    void appendIntegrationParticle(float4 v, float ro, uint iterations)
    {
        for (int i = 0; i < iterations; i++)
        {
            V.push_back(v.x);
            V.push_back(v.y);
            V.push_back(v.z);
            V.push_back(v.w);

            ros.push_back(ro);
            numNeighbors.push_back(0);
            lambda.push_back(0.f);
        }
        neighbors.resize(V.size() * MAX_FLUID_NEIGHBORS);
    }

    void freeIntegrationVectors()
    {
         V.clear();
         lambda.clear();
         denom.clear();
         ros.clear();
         neighbors.clear();
         numNeighbors.clear();

         V.shrink_to_fit();
         lambda.shrink_to_fit();
         denom.shrink_to_fit();
         ros.shrink_to_fit();
         neighbors.shrink_to_fit();
         numNeighbors.shrink_to_fit();

         checkCudaErrors(hiprandDestroyGenerator(gen));
         freeArray(rands);
    }

    void setParameters(SimParams *hostParams)
    {
        // copy parameters to constant memory
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)));
    }

















    /*****************************************************************************
     *                              UPDATE POSITIONS
     *****************************************************************************/

    void integrateSystem(float *pos, float deltaTime, uint numParticles)
    {
        thrust::device_ptr<float4> d_pos4((float4 *)pos);
        thrust::device_ptr<float4> d_vel4((float4 *)thrust::raw_pointer_cast(V.data()));

        // copy current positions for reference later
        copyToXstar(pos, numParticles);

        // guess new positions based on forces
        thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_vel4)),
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4+numParticles, d_vel4+numParticles)),
            integrate_functor(deltaTime));
    }





















    /*****************************************************************************
     *                              BUILD GRID
     *****************************************************************************/

    void calcHash(uint *gridParticleHash, uint *gridParticleIndex, float *pos, int numParticles)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // execute the kernel
        calcHashD<<< numBlocks, numThreads >>>(gridParticleHash, gridParticleIndex, (float4 *) pos, numParticles);

#ifdef PRINT
        printf("HASHES:\n");
        thrust::device_ptr<uint> dGPH(gridParticleHash);
        for (uint i = 0; i < numParticles; i++)
        {
            printf("particle: %u: hash: %u\n", i, (uint)*(dGPH + i));
        }
        printf("\n");
#endif

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }


    void reorderDataAndFindCellStart(uint  *cellStart,
                                     uint  *cellEnd,
                                     float *sortedPos,
                                     float *sortedW,
                                     int   *sortedPhase,
                                     uint  *gridParticleHash,
                                     uint  *gridParticleIndex,
                                     float *oldPos,
                                     uint   numParticles,
                                     uint   numCells)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // set all cells to empty
        checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

        float *dW = getWRawPtr();
        int *dPhase = getPhaseRawPtr();

        checkCudaErrors(hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, invMassTex, dW, numParticles*sizeof(float)));
        checkCudaErrors(hipBindTexture(0, oldPhaseTex, dPhase, numParticles*sizeof(int)));

        uint smemSize = sizeof(uint)*(numThreads+1);
        reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(cellStart,
                                                                           cellEnd,
                                                                           (float4 *) sortedPos,
                                                                           sortedW,
                                                                           sortedPhase,
                                                                           gridParticleHash,
                                                                           gridParticleIndex,
                                                                           (float4 *) oldPos,
                                                                           dW,
                                                                           dPhase,
                                                                           numParticles);
        getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");

#ifdef PRINT
        printf("Sorted:\n");
        thrust::device_ptr<uint> dGPH(gridParticleHash);
        thrust::device_ptr<uint> dGPI(gridParticleIndex);
        for (uint i = 0; i < numParticles; i++)
        {
            printf("i: %u: hash: %u\n", i, (uint)*(dGPH + i));
            printf("i: %u: part: %u\n", i, (uint)*(dGPI + i));
        }
        printf("\n");


        printf("Sorted:\n");
        thrust::device_ptr<uint> dstart(cellStart);
        thrust::device_ptr<uint> dend(cellEnd);
        for (uint i = 0; i < 16; i++)
        {
            printf("i: %u: start: %u\n", i, (uint)*(dstart + i));
            printf("i: %u: end: %u\n", i, (uint)*(dend + i));
        }
        printf("\n");
#endif

        checkCudaErrors(hipUnbindTexture(oldPosTex));
        checkCudaErrors(hipUnbindTexture(invMassTex));
        checkCudaErrors(hipUnbindTexture(oldPhaseTex));
    }

    void sortParticles(uint *dGridParticleHash, uint *dGridParticleIndex, uint numParticles)
    {
        thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),
                            thrust::device_ptr<uint>(dGridParticleHash + numParticles),
                            thrust::device_ptr<uint>(dGridParticleIndex));
    }


































    /*****************************************************************************
     *                              PROCESS COLLISIONS
     *****************************************************************************/

    void sortByType(float *dPos, uint numParticles)
    {

    }

    void collideWorld(float *pos, float *sortedPos, uint numParticles, int3 minBounds, int3 maxBounds)
    {
        thrust::device_ptr<float4> d_pos4((float4 *)pos);
        thrust::device_ptr<float4> d_Xstar((float4*)getXstarRawPtr());
        thrust::device_ptr<int> d_phase(getPhaseRawPtr());

        // create random vars for boundary collisions
        checkCudaErrors(hiprandGenerateUniform(gen, rands, 6));

        // check for boundary collisions and move particles
//        thrust::for_each
//        thrust::transform(d_pos4, d_pos4 + numParticles, d_Xstar, d_pos4, collide_world_functor(rands, minBounds, maxBounds));

        thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_Xstar, d_phase)),
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4+numParticles, d_Xstar+numParticles, d_phase+numParticles)),
            collide_world_functor(rands, minBounds, maxBounds));
    }

    void collide(float *particles,
                 float *sortedPos,
                 float *sortedW,
                 int   *sortedPhase,
                 uint  *gridParticleIndex,
                 uint  *cellStart,
                 uint  *cellEnd,
                 uint   numParticles,
                 uint   numCells)
    {
        checkCudaErrors(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, invMassTex, sortedW, numParticles*sizeof(float)));
        checkCudaErrors(hipBindTexture(0, oldPhaseTex, sortedPhase, numParticles*sizeof(int)));

        checkCudaErrors(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
        checkCudaErrors(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));

        // store neighbors
        uint *dNeighbors = thrust::raw_pointer_cast(neighbors.data());
        uint *dNumNeighbors = thrust::raw_pointer_cast(numNeighbors.data());
        float *dXstar = getXstarRawPtr();

        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 64, numBlocks, numThreads);

        // execute the kernel
        collideD<<< numBlocks, numThreads >>>((float4 *)particles,
                                              (float4 *)dXstar,
                                              (float4 *)sortedPos,
                                              sortedW,
                                              sortedPhase,
                                              gridParticleIndex,
                                              cellStart,
                                              cellEnd,
                                              numParticles,
                                              dNeighbors,
                                              dNumNeighbors);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");

        checkCudaErrors(hipUnbindTexture(oldPosTex));
        checkCudaErrors(hipUnbindTexture(invMassTex));
        checkCudaErrors(hipUnbindTexture(oldPhaseTex));

        checkCudaErrors(hipUnbindTexture(cellStartTex));
        checkCudaErrors(hipUnbindTexture(cellEndTex));
    }

























    /*****************************************************************************
     *                              UPDATE VELOCITIES
     *****************************************************************************/

    void calcVelocity(float *dpos, float deltaTime, uint numParticles)
    {
        float *dXstar = getXstarRawPtr();
        thrust::device_ptr<float4> d_Xstar((float4*)dXstar);
        thrust::device_ptr<float4> d_pos((float4*)dpos);
        thrust::device_ptr<float4> d_vel((float4*)thrust::raw_pointer_cast(V.data()));


        thrust::transform(d_pos, d_pos + numParticles, d_Xstar, d_vel, subtract_functor(deltaTime));

    }























    /*****************************************************************************
     *                              SOLVE FLUIDS
     *****************************************************************************/
    void solveFluids(float *sortedPos,
                     int   *sortedPhase,
                     uint  *gridParticleIndex,
                     uint  *cellStart,
                     uint  *cellEnd,
                     float *particles,
                     uint   numParticles,
                     uint   numCells,
                     float4 mousePos)
    {
        checkCudaErrors(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, oldPhaseTex, sortedPhase, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
        checkCudaErrors(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));

        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        float *dLambda = thrust::raw_pointer_cast(lambda.data());
//        float *dDenom = thrust::raw_pointer_cast(denom.data());
        uint *dNeighbors = thrust::raw_pointer_cast(neighbors.data());
        uint *dNumNeighbors = thrust::raw_pointer_cast(numNeighbors.data());
        float *dRos = thrust::raw_pointer_cast(ros.data());

//        printf("ros: %u, numParts: %u\n", (uint)ros.size(), numParticles);

        // execute the kernel
        findLambdasD<<< numBlocks, numThreads >>>(dLambda,
                                                  (float4 *)sortedPos,
                                                  gridParticleIndex,
                                                  cellStart,
                                                  cellEnd,
                                                  numParticles,
                                                  dNeighbors,
                                                  dNumNeighbors,
                                                  dRos);

        // execute the kernel
        solveFluidsD<<< numBlocks, numThreads >>>(dLambda,
                                                  (float4 *)sortedPos,
                                                  gridParticleIndex,
                                                  (float4 *) particles,
                                                  numParticles,
                                                  dNeighbors,
                                                  dNumNeighbors,
                                                  dRos);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");

        checkCudaErrors(hipUnbindTexture(oldPosTex));
        checkCudaErrors(hipUnbindTexture(oldPhaseTex));
        checkCudaErrors(hipUnbindTexture(cellStartTex));
        checkCudaErrors(hipUnbindTexture(cellEndTex));
    }
}
