#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hiprand.h>
#include <stdio.h>

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>

#include "hip/hip_runtime_api.h"
#include "integration_kernel.cuh"
#include "util.cuh"
#include "shared_variables.cuh"

//#define PRINT

hiprandGenerator_t gen(0);

thrust::device_vector<float> prev; // previous positions
thrust::device_vector<float> lambda;
thrust::device_vector<float> denom;

thrust::device_vector<float> ros;

thrust::device_vector<uint> neighbors;
thrust::device_vector<uint> numNeighbors;

thrust::device_vector<float> tris;
thrust::device_vector<uint> trisGroup;
thrust::device_vector<float> centRad;

float *rands;

extern "C"
{
    /*****************************************************************************
     *                              INITIALIZATION
     *****************************************************************************/


    void initIntegration()
    {
        allocateArray((void **)&rands, 6 * sizeof(float));
        checkCudaErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
        checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
    }

    void appendIntegrationParticle(float4 pos, float ro, uint iterations)
    {
        for (int i = 0; i < iterations; i++)
        {
            prev.push_back(pos.x);
            prev.push_back(pos.y);
            prev.push_back(pos.z);
            prev.push_back(pos.w);

            ros.push_back(ro);
            numNeighbors.push_back(0);
            lambda.push_back(0.f);
        }
        neighbors.resize(prev.size() * MAX_FLUID_NEIGHBORS);
    }

    void addTriGroup(uint start, uint end, float3 center, float radius)
    {
        trisGroup.push_back(start);
        trisGroup.push_back(end);
        centRad.push_back(center.x);
        centRad.push_back(center.y);
        centRad.push_back(center.z);
        centRad.push_back(radius);
    }

    void addTriangle(float3 a, float3 b, float3 c, float3 n)
    {
        tris.push_back(a.x);
        tris.push_back(a.y);
        tris.push_back(a.z);
        tris.push_back(n.x);
        tris.push_back(b.x);
        tris.push_back(b.y);
        tris.push_back(b.z);
        tris.push_back(n.y);
        tris.push_back(c.x);
        tris.push_back(c.y);
        tris.push_back(c.z);
        tris.push_back(n.z);
    }

    void freeIntegrationVectors()
    {
         prev.clear();
         lambda.clear();
         denom.clear();
         ros.clear();
         neighbors.clear();
         numNeighbors.clear();
         tris.clear();
         trisGroup.clear();
         centRad.clear();

         prev.shrink_to_fit();
         lambda.shrink_to_fit();
         denom.shrink_to_fit();
         ros.shrink_to_fit();
         neighbors.shrink_to_fit();
         numNeighbors.shrink_to_fit();
         tris.shrink_to_fit();
         trisGroup.shrink_to_fit();
         centRad.shrink_to_fit();

         checkCudaErrors(hiprandDestroyGenerator(gen));
         freeArray(rands);
    }

    void setParameters(SimParams *hostParams)
    {
        // copy parameters to constant memory
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)));
    }

















    /*****************************************************************************
     *                              UPDATE POSITIONS
     *****************************************************************************/

    void integrateSystem(float *pos, float prevTime, float deltaTime, uint numParticles)
    {
        thrust::device_ptr<float4> d_pos4((float4*)pos);
        thrust::device_ptr<float4> d_prev4((float4*)thrust::raw_pointer_cast(prev.data()));

        // copy current positions for reference later
//        copyToXstar(pos, numParticles);

        // guess new positions based on forces
        thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_prev4)),
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4+numParticles, d_prev4+numParticles)),
            integrate_functor(prevTime, deltaTime));
    }





















    /*****************************************************************************
     *                              BUILD GRID
     *****************************************************************************/

    void calcHash(uint *gridParticleHash, uint *gridParticleIndex, float *pos, int numParticles)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // execute the kernel
        calcHashD<<< numBlocks, numThreads >>>(gridParticleHash, gridParticleIndex, (float4 *) pos, numParticles);

#ifdef PRINT
        printf("HASHES:\n");
        thrust::device_ptr<uint> dGPH(gridParticleHash);
        for (uint i = 0; i < numParticles; i++)
        {
            printf("particle: %u: hash: %u\n", i, (uint)*(dGPH + i));
        }
        printf("\n");
#endif

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }


    void reorderDataAndFindCellStart(uint  *cellStart,
                                     uint  *cellEnd,
                                     float *sortedPos,
                                     float *sortedW,
                                     int   *sortedPhase,
                                     uint  *gridParticleHash,
                                     uint  *gridParticleIndex,
                                     float *oldPos,
                                     uint   numParticles,
                                     uint   numCells)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // set all cells to empty
        checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

        float *dW = getWRawPtr();
        int *dPhase = getPhaseRawPtr();

        checkCudaErrors(hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, invMassTex, dW, numParticles*sizeof(float)));
        checkCudaErrors(hipBindTexture(0, oldPhaseTex, dPhase, numParticles*sizeof(int)));

        uint smemSize = sizeof(uint)*(numThreads+1);
        reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(cellStart,
                                                                           cellEnd,
                                                                           (float4 *) sortedPos,
                                                                           sortedW,
                                                                           sortedPhase,
                                                                           gridParticleHash,
                                                                           gridParticleIndex,
                                                                           (float4 *) oldPos,
                                                                           dW,
                                                                           dPhase,
                                                                           numParticles);
        getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");

#ifdef PRINT
        printf("Sorted:\n");
        thrust::device_ptr<uint> dGPH(gridParticleHash);
        thrust::device_ptr<uint> dGPI(gridParticleIndex);
        for (uint i = 0; i < numParticles; i++)
        {
            printf("i: %u: hash: %u\n", i, (uint)*(dGPH + i));
            printf("i: %u: part: %u\n", i, (uint)*(dGPI + i));
        }
        printf("\n");


        printf("Sorted:\n");
        thrust::device_ptr<uint> dstart(cellStart);
        thrust::device_ptr<uint> dend(cellEnd);
        for (uint i = 0; i < 16; i++)
        {
            printf("i: %u: start: %u\n", i, (uint)*(dstart + i));
            printf("i: %u: end: %u\n", i, (uint)*(dend + i));
        }
        printf("\n");
#endif

        checkCudaErrors(hipUnbindTexture(oldPosTex));
        checkCudaErrors(hipUnbindTexture(invMassTex));
        checkCudaErrors(hipUnbindTexture(oldPhaseTex));
    }

    void sortParticles(uint *dGridParticleHash, uint *dGridParticleIndex, uint numParticles)
    {
        thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),
                            thrust::device_ptr<uint>(dGridParticleHash + numParticles),
                            thrust::device_ptr<uint>(dGridParticleIndex));
    }


































    /*****************************************************************************
     *                              PROCESS COLLISIONS
     *****************************************************************************/

    void sortByType(float *dPos, uint numParticles)
    {

    }

    void collideWorld(float *pos, float *sortedPos, uint numParticles, float3 playerPos, float playerRadius, int3 minBounds, int3 maxBounds)
    {
        thrust::device_ptr<float4> d_pos4((float4 *)pos);
        thrust::device_ptr<float4> d_prev4((float4*)thrust::raw_pointer_cast(prev.data()));
        thrust::device_ptr<int> d_phase(getPhaseRawPtr());

        // create random vars for boundary collisions
        checkCudaErrors(hiprandGenerateUniform(gen, rands, 6));

        // check for boundary collisions and move particles
//        thrust::for_each
//        thrust::transform(d_pos4, d_pos4 + numParticles, d_Xstar, d_pos4, collide_world_functor(rands, minBounds, maxBounds));

        thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_prev4, d_phase)),
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4+numParticles, d_prev4+numParticles, d_phase+numParticles)),
            collide_world_functor(rands, minBounds, maxBounds));
    }

    void collide(float *particles,
                 float *sortedPos,
                 float *sortedW,
                 int   *sortedPhase,
                 uint  *gridParticleIndex,
                 uint  *cellStart,
                 uint  *cellEnd,
                 uint   numParticles,
                 uint   numCells,
                 float3 playerPos,
                 float  playerRadius)
    {
        checkCudaErrors(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, invMassTex, sortedW, numParticles*sizeof(float)));
        checkCudaErrors(hipBindTexture(0, oldPhaseTex, sortedPhase, numParticles*sizeof(int)));

        checkCudaErrors(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
        checkCudaErrors(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));

        // store neighbors
        uint *dNeighbors = thrust::raw_pointer_cast(neighbors.data());
        uint *dNumNeighbors = thrust::raw_pointer_cast(numNeighbors.data());
        float *dPrev = thrust::raw_pointer_cast(prev.data());
        float4 *dTris = (float4*) thrust::raw_pointer_cast(tris.data());
        uint2 *dTrisGroup = (uint2*) thrust::raw_pointer_cast(trisGroup.data());
        float4 *dCentRad = (float4*) thrust::raw_pointer_cast(centRad.data());

        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 64, numBlocks, numThreads);

        // execute the kernel
        collideD<<< numBlocks, numThreads >>>((float4 *)particles,
                                              (float4 *)dPrev,
                                              gridParticleIndex,
                                              cellStart,
                                              cellEnd,
                                              numParticles,
                                              dNeighbors,
                                              dNumNeighbors,
                                              playerPos,
                                              playerRadius,
                                              dTris,
                                              dTrisGroup,
                                              dCentRad,
                                              trisGroup.size() / 2);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");

        checkCudaErrors(hipUnbindTexture(oldPosTex));
        checkCudaErrors(hipUnbindTexture(invMassTex));
        checkCudaErrors(hipUnbindTexture(oldPhaseTex));

        checkCudaErrors(hipUnbindTexture(cellStartTex));
        checkCudaErrors(hipUnbindTexture(cellEndTex));
    }

























    /*****************************************************************************
     *                              UPDATE VELOCITIES
     *****************************************************************************/

//    void calcVelocity(float *dpos, float deltaTime, uint numParticles)
//    {
//        float *dXstar = getXstarRawPtr();
//        thrust::device_ptr<float4> d_Xstar((float4*)dXstar);
//        thrust::device_ptr<float4> d_pos((float4*)dpos);
//        thrust::device_ptr<float4> d_vel((float4*)thrust::raw_pointer_cast(V.data()));


//        thrust::transform(d_pos, d_pos + numParticles, d_Xstar, d_vel, subtract_functor(deltaTime));

//    }























    /*****************************************************************************
     *                              SOLVE FLUIDS
     *****************************************************************************/
    void solveFluids(float *sortedPos,
                     int   *sortedPhase,
                     uint  *gridParticleIndex,
                     uint  *cellStart,
                     uint  *cellEnd,
                     float *particles,
                     uint   numParticles,
                     uint   numCells,
                     float4 mousePos)
    {
        checkCudaErrors(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, oldPhaseTex, sortedPhase, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
        checkCudaErrors(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));

        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        float *dLambda = thrust::raw_pointer_cast(lambda.data());
//        float *dDenom = thrust::raw_pointer_cast(denom.data());
        uint *dNeighbors = thrust::raw_pointer_cast(neighbors.data());
        uint *dNumNeighbors = thrust::raw_pointer_cast(numNeighbors.data());
        float *dRos = thrust::raw_pointer_cast(ros.data());

//        printf("ros: %u, numParts: %u\n", (uint)ros.size(), numParticles);

        // execute the kernel
        findLambdasD<<< numBlocks, numThreads >>>(dLambda,
                                                  (float4 *)sortedPos,
                                                  gridParticleIndex,
                                                  cellStart,
                                                  cellEnd,
                                                  numParticles,
                                                  dNeighbors,
                                                  dNumNeighbors,
                                                  dRos);

        // execute the kernel
        solveFluidsD<<< numBlocks, numThreads >>>(dLambda,
                                                  (float4 *)sortedPos,
                                                  gridParticleIndex,
                                                  (float4 *) particles,
                                                  numParticles,
                                                  dNeighbors,
                                                  dNumNeighbors,
                                                  dRos);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");

        checkCudaErrors(hipUnbindTexture(oldPosTex));
        checkCudaErrors(hipUnbindTexture(oldPhaseTex));
        checkCudaErrors(hipUnbindTexture(cellStartTex));
        checkCudaErrors(hipUnbindTexture(cellEndTex));
    }
}
